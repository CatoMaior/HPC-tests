#define LEN_ARR 100000000
#define BLOCK_SIZE 500
#define N_BLOCKS 500


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void sumVector(float *a, float *b, float *result, int n) {
    int firstIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = firstIndex; i < n; i += BLOCK_SIZE*N_BLOCKS) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    float *a = (float*)malloc(sizeof(float)*LEN_ARR);
    float *b = (float*)malloc(sizeof(float)*LEN_ARR);
    float *result = (float*)malloc(sizeof(float)*LEN_ARR);
    float *d_a, *d_b, *d_result;

    for(int i = 0; i < LEN_ARR; i++){
        a[i] = 30.0f;
        b[i] = 12.0f;
    }

    hipMalloc((void **) &d_a, sizeof(float)*LEN_ARR);
    hipMalloc((void **) &d_b, sizeof(float)*LEN_ARR);
    hipMalloc((void **) &d_result, sizeof(float)*LEN_ARR);

    hipMemcpy(d_a, a, sizeof(float)*LEN_ARR, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*LEN_ARR, hipMemcpyHostToDevice);

    sumVector<<<N_BLOCKS, BLOCK_SIZE>>>(d_a, d_b, d_result, LEN_ARR);

    hipMemcpy(result, d_result, sizeof(float)*LEN_ARR, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    free(a); 
    free(b); 
    free(result);

    return 0;
}

