#define LEN_ARR 100000000
#define BLOCK_SIZE 500
#define N_BLOCKS 500


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void sumVector(float *a, float *b, float *result, int n) {
    int firstIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = firstIndex; i < n; i += BLOCK_SIZE * N_BLOCKS) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    float *a = (float*)malloc(sizeof(float) * LEN_ARR);
    float *b = (float*)malloc(sizeof(float) * LEN_ARR);
    float *result = (float*)malloc(sizeof(float) * LEN_ARR);
    float *gpuA, *gpuB, *gpuResult;

    for(int i = 0; i < LEN_ARR; i++){
        a[i] = 30.0f;
        b[i] = 12.0f;
    }

    hipMalloc((void **) &gpuA, sizeof(float) * LEN_ARR);
    hipMalloc((void **) &gpuB, sizeof(float) * LEN_ARR);
    hipMalloc((void **) &gpuResult, sizeof(float) * LEN_ARR);

    hipMemcpy(gpuA, a, sizeof(float) * LEN_ARR, hipMemcpyHostToDevice);
    hipMemcpy(gpuB, b, sizeof(float) * LEN_ARR, hipMemcpyHostToDevice);

    sumVector<<<N_BLOCKS, BLOCK_SIZE>>>(gpuA, gpuB, gpuResult, LEN_ARR);

    hipMemcpy(result, gpuResult, sizeof(float) * LEN_ARR, hipMemcpyDeviceToHost);

    hipFree(gpuA);
    hipFree(gpuB);
    hipFree(gpuResult);

    free(a); 
    free(b); 
    free(result);

    return 0;
}

